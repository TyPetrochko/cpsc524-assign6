#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;
  
  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
      c[index] += a[indexa]*b[indexb];
  }
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {
  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
        cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}

int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, m, p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<6) || (argc>7)) {
    printf("Usage: rectangle <n> <m> <p> <block dim> <grid dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim = atoi(argv[5]); // Square grid
  if (Grid_Dim*Block_Dim < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  if (argc==7) {
    gpunum = atoi(argv[6]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = %d %d %d\n",n, m, p);
  printf("Block_Dim = %d, Grid_Dim = %d\n",Block_Dim,Grid_Dim);

  dim3 Grid(Grid_Dim, Grid_Dim); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  size_a = n * p * sizeof(FP); // number of bytes in total in arrays
  size_b = m * p * sizeof(FP); // number of bytes in total in arrays
  size_c = m * n * sizeof(FP); // number of bytes in total in arrays

  a = (FP*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(size_b);
  c = (FP*) malloc(size_c); // results from GPU

  srand(12345);
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,m,p);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  cpu_matrixmult(a,b,c,n,m,p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, sumc, ci;
  sumc = 0;
  for(i=0;i < m*n;i++) {
    ci = (double) c[i];
    sumc += ci*ci;
  }
  sumc = sqrt(sumc);
  error = sumc;
  printf("Total error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
