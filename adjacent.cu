#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_WIDTH 16
#define ADJ_TILES 2
#define BUF_SIZE = (ADJ_TILES * BLOCK_WIDTH * BLOCK_WIDTH * sizeof(FP))

#define DEBUG false

void globalDebugMatrix(int m, int n, FP *matrix){
  for(int i = 0; i < n; i++){
    for(int j = 0; j < m; j++){
      printf("%e ", matrix[m * i + j]);
    }

    printf("\n");
  }
}

__device__ void debugMatrix(int m, int n, FP *matrix){
  for(int i = 0; i < n; i++){
    for(int j = 0; j < m; j++){
      printf("%e ", matrix[m * i + j]);
    }

    printf("\n");
  }
}

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {
  if(blockDim.x != blockDim.y){
    printf("Error - block is not square!\n");
    return;
  }

  int block_width = blockDim.x;
  
  int threadx = threadIdx.x;
  int thready = threadIdx.y;
  int blockx = blockIdx.x;
  int blocky = blockIdx.y;

  int xcoord = blockx*block_width + threadx;
  int ycoord = blocky*block_width + thready;
  

  if(xcoord > m || ycoord > n){
    printf("We're not needed!\n"); // tbh surprised we can call printf from device
    return;
  }

  __shared__ FP As_buf[ADJ_TILES * BLOCK_WIDTH * BLOCK_WIDTH * sizeof(FP)];
  __shared__ FP Bs_buf[ADJ_TILES * BLOCK_WIDTH * BLOCK_WIDTH * sizeof(FP)];

  FP *As, *Bs;

  FP c_value = 0.;
  for(int i = 0; i < (p / block_width); i++){

    // every ADJ_TILES iterations, recalculate the buffers!
    if(i % ADJ_TILES == 0){
      As = As_buf;
      Bs = Bs_buf;
      
      __syncthreads();
      
      for(int q = 0; q < ADJ_TILES; q++){
        int a_y = blocky*block_width + thready;
        int a_x = (q + i)*block_width + threadx;

        int b_y = (q + i)*block_width + thready;
        int b_x = blockx*block_width + threadx;

        // each thread computes one matrix value
        As[block_width * thready + threadx] = a[p * a_y + a_x];
        Bs[block_width * thready + threadx] = b[m * b_y + b_x];

        As += BLOCK_WIDTH*BLOCK_WIDTH*sizeof(FP);
        Bs += BLOCK_WIDTH*BLOCK_WIDTH*sizeof(FP);
      }

      As = As_buf;
      Bs = Bs_buf;

      __syncthreads();
    }else{
        As += BLOCK_WIDTH*BLOCK_WIDTH*sizeof(FP);
        Bs += BLOCK_WIDTH*BLOCK_WIDTH*sizeof(FP);
    }
    
    for(int e = 0; e < block_width; e++){
      c_value += As[thready * block_width + e] * Bs[e * block_width + threadx];
    }
  }

  c[m*ycoord + xcoord] = c_value;
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {
  // Taken directly from slides
  for(int k = 0; k < p; k++){
    for(int i = 0; i < n; i++){
      FP r = a[(i * p) + k];
      int cbase = i * m;
      int bbase = k * m;
      for(int j = 0; j < m; j++){
        c[cbase + j] -= r * b[bbase + j];
      }
    }
  }
}

int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, m, p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<6) || (argc>7)) {
    printf("Usage: tiled <n> <m> <p> <block dim> <grid dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim = atoi(argv[5]); // Square grid
  if (Grid_Dim*Block_Dim < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  if (argc==7) {
    gpunum = atoi(argv[6]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = %d %d %d\n",n, m, p);
  printf("Block_Dim = %d, Grid_Dim = %d\n",Block_Dim,Grid_Dim);

  dim3 Grid(Grid_Dim, Grid_Dim); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  size_a = n * p * sizeof(FP); // number of bytes in total in arrays
  size_b = m * p * sizeof(FP); // number of bytes in total in arrays
  size_c = m * n * sizeof(FP); // number of bytes in total in arrays

  a = (FP*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(size_b);
  c = (FP*) malloc(size_c); // results from GPU

  srand(12345);
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
            // a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
            // b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  // gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,m,p);
  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,m,p);
  // printf("Allocating %d bytes total\n", 2*Block_Dim*Block_Dim*sizeof(FP));

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  if(DEBUG == false) goto cleanup;
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  cpu_matrixmult(a,b,c,n,m,p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, sumc, ci;
  sumc = 0;
  for(i=0;i < m*n;i++) {
    ci = (double) c[i];
    sumc += ci*ci;
    // printf("Error at %d is %e\n", i, ci);

    if(ci > 0.01 || ci < -0.01)
      printf("Error at row %d, column %d, is %e\n", i / m, i % m, ci);
  }
  sumc = sqrt(sumc);
  error = sumc;
  printf("Total error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------
cleanup:
  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
