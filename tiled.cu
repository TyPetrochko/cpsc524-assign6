#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_WIDTH 16

// N rows (height)
// M columns (width)

void globalDebugMatrix(int m, int n, FP *matrix){
  for(int i = 0; i < n; i++){
    for(int j = 0; j < m; j++){
      printf("%e ", matrix[m * i + j]);
    }

    printf("\n");
  }
}

__device__ void debugMatrix(int m, int n, FP *matrix){
  for(int i = 0; i < n; i++){
    for(int j = 0; j < m; j++){
      printf("%e ", matrix[m * i + j]);
    }

    printf("\n");
  }
}

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {

  // int col = threadIdx.x + blockDim.x * blockIdx.x;
  // int row = threadIdx.y + blockDim.y * blockIdx.y;

  // int indexb = col;
  // int index = row * m + col;
  // 
  // if(col < m && row < n) {
  //   c[index] = 0.;
  //   for (int indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
  //     c[index] += a[indexa]*b[indexb];
  // }

  if(blockDim.x != blockDim.y){
    printf("Error - block is not square!\n");
    return;
  }

  int debug = false;

  int block_width = blockDim.x;
  
  int threadx = threadIdx.x;
  int thready = threadIdx.y;
  int blockx = blockIdx.x;
  int blocky = blockIdx.y;

  if(threadx == 0 && thready == 0 && blockx == 1 && blocky == 0) debug = false;
  
  int xcoord = blockx*block_width + threadx;
  int ycoord = blocky*block_width + thready;
  

  if(xcoord > m || ycoord > n){
    printf("We're not needed!\n"); // tbh surprised we can call printf from device
    return;
  }
  

  // for now just do perfect matches
  if(p % block_width > 0.0)
    printf("WARNING: matrix p dimension is not a perfect multiple of block width!\n");
  if(m % block_width > 0.0)
    printf("WARNING: matrix m dimension is not a perfect multiple of block width!\n");
  if(n % block_width > 0.0)
    printf("WARNING: matrix n dimension is not a perfect multiple of block width!\n");
    
  // extern __shared__ FP As[];
  // FP *Bs = As + (block_width * block_width * sizeof(FP));

  __shared__ FP As[BLOCK_WIDTH * BLOCK_WIDTH * sizeof(FP)];
  __shared__ FP Bs[BLOCK_WIDTH * BLOCK_WIDTH * sizeof(FP)];

  FP c_value = 0.;
  for(int i = 0; i < (p / block_width); i++){

    if(debug){
      printf("On iteration %d\n", i);
    }
    

    // __shared__ FP *As = cudaMalloc(sizeof(FP) * block_width * block_width);
    // __shared__ FP *Bs = cudaMalloc(sizeof(FP) * block_width * block_width);

    int a_y = blocky*block_width + thready;
    int a_x = i*block_width + threadx;

    int b_y = i*block_width + thready;
    int b_x = blockx*block_width + threadx;

    // each thread computes one matrix value
    As[block_width * thready + threadx] = a[p * a_y + a_x];
    Bs[block_width * thready + threadx] = b[m * b_y + b_x];
    if(debug)printf("My copied vals in a and b are %e %e\n", As[block_width * thready + threadx], Bs[block_width * thready + threadx]);

    if(debug){
      As[block_width * thready + threadx] = a[p * a_y + a_x];
      Bs[block_width * thready + threadx] = b[m * b_y + b_x];
      printf("My copied vals in a and b are %e %e\n", As[block_width * thready + threadx], Bs[block_width * thready + threadx]);
    }
    
    // wait for all to finish computing As, Bs
    __syncthreads();
    
    if(debug){
      printf("My ax, ay, bx, by are %d %d %d %d\n", a_x, a_y, b_x, b_y);
      printf("My copied vals in a and b are %e %e\n", As[block_width * thready + threadx], Bs[block_width * thready + threadx]);
      printf("As:\n");
      debugMatrix(block_width, block_width, As);
      printf("Bs:\n");
      debugMatrix(block_width, block_width, Bs);
      printf("\n\n");
    }


    for(int e = 0; e < block_width; e++){
      c_value += As[thready * block_width + e] * Bs[e * block_width + threadx];
    }

    // let other threads finish before computing next As, Bs
    __syncthreads();
  }

  c[m*ycoord + xcoord] = c_value;
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {
  // Taken directly from slides
  printf("Broken version first!\n");
  for(int k = 0; k < p; k++){
    for(int i = 0; i < n; i++){
      FP r = a[(i * p) + k];
      int cbase = i * m;
      int bbase = k * m;
      for(int j = 0; j < m; j++){
        c[cbase + j] -= r * b[bbase + j];
      }
    }
  }
}

int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, m, p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<6) || (argc>7)) {
    printf("Usage: tiled <n> <m> <p> <block dim> <grid dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim = atoi(argv[5]); // Square grid
  if (Grid_Dim*Block_Dim < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  if (argc==7) {
    gpunum = atoi(argv[6]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = %d %d %d\n",n, m, p);
  printf("Block_Dim = %d, Grid_Dim = %d\n",Block_Dim,Grid_Dim);

  dim3 Grid(Grid_Dim, Grid_Dim); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  size_a = n * p * sizeof(FP); // number of bytes in total in arrays
  size_b = m * p * sizeof(FP); // number of bytes in total in arrays
  size_c = m * n * sizeof(FP); // number of bytes in total in arrays

  a = (FP*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(size_b);
  c = (FP*) malloc(size_c); // results from GPU

  srand(12345);
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //       a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // printf("A:\n");
  // globalDebugMatrix(p, n, a);

  // printf("B:\n");
  // globalDebugMatrix(m, p, b);
  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  // gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,m,p);
  gpu_matrixmult<<<Grid,Block, 2*Block_Dim*Block_Dim*sizeof(FP)>>>(dev_a,dev_b,dev_c,n,m,p);
  printf("Allocating %d bytes total\n", 2*Block_Dim*Block_Dim*sizeof(FP));

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  cpu_matrixmult(a,b,c,n,m,p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, sumc, ci;
  sumc = 0;
  for(i=0;i < m*n;i++) {
    ci = (double) c[i];
    sumc += ci*ci;
  }
  sumc = sqrt(sumc);
  error = sumc;
  printf("Total error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
